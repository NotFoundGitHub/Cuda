#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include <windows.h>


#define NThreads 512
#define NBlocks 1

#define Num NThreads*NBlocks




__device__ void swap(int &a, int &b){
	int t = a;
	a = b;
	b = t;
}

__global__ void bitonic_sort(int* arr){
	extern __shared__ int shared_arr[];
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//const unsigned int tid = threadIdx.x;
	shared_arr[tid] = arr[tid];
	__syncthreads();

	//for(int i=2; i<=blociDim.x; i<<=1){
	for(unsigned int i=2; i<=Num; i<<=1){
		for(unsigned int j=i>>1; j>0; j>>=1){
			unsigned int tid_comp = tid ^ j;
			if(tid_comp > tid){
				if((tid & i)==0){ //ascending
					if(shared_arr[tid]>shared_arr[tid_comp]){
						swap(shared_arr[tid],shared_arr[tid_comp]);
					}
				}
				else{ //desending
					if(shared_arr[tid]<shared_arr[tid_comp]){
						swap(shared_arr[tid],shared_arr[tid_comp]);
					}
				}
			}
			__syncthreads();
		}
	}
	arr[tid] = shared_arr[tid];
}


void swap(int s[],int i,int j)
{
	int temp;
	temp=s[i];
	s[i]=s[j];
	s[j]=temp;
}


void QuickSort(int s[],int low,int high)
{
	int i;
	int last;       
	if(low<high)    
	{
		last=low;   

		for(i=low+1;i<=high;i++)
		{
			if(s[i]<s[low])
				swap(s,++last,i);
		}

		swap(s,last,low);
		QuickSort(s,low,last-1); 
		QuickSort(s,last+1,high);
	}
}


int main(int argc, char* argv[])
{

	int* arr= (int*) malloc(Num*sizeof(int));
	int* arr1= (int*) malloc(Num*sizeof(int));
	//init array value
	time_t t;
	clock_t start1,end1;
	double usetime;


	srand((unsigned)time(&t));
	for(int i=0;i<Num;i++){
		arr[i] = rand() % 1000; 
	}

	//init device variable
	int* ptr;
	hipMalloc((void**)&ptr,Num*sizeof(int));
	hipMemcpy(ptr,arr,Num*sizeof(int),hipMemcpyHostToDevice);

	for(int i=0;i<Num;i++){
		printf("%d\t",arr[i]);
	}


	printf("\n---------------- init ----------------\n");

	LARGE_INTEGER nFreq;
	 LARGE_INTEGER nBeginTime;

     LARGE_INTEGER nEndTime;
	 double utime;

	 QueryPerformanceFrequency(&nFreq);

     QueryPerformanceCounter(&nBeginTime); 

	start1 = clock();

	for(int i=0;i<Num;i++){
		arr1[i]=arr[i];

	}


	

		hipEvent_t start, stop;
		float elapsedTime = 0.0;

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);


		dim3 blocks(NBlocks,1);
		dim3 threads(NThreads,1);


		//bitonic_sort<<<blocks,threads,Num*sizeof(int)>>>(ptr);
		bitonic_sort<<<blocks,threads,Num*sizeof(int)>>>(ptr);

		//bitonic_sort<<<1,Num,Num*sizeof(int)>>>(ptr);


		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsedTime, start, stop);

		hipMemcpy(arr,ptr,Num*sizeof(int),hipMemcpyDeviceToHost);

		printf("\n---------------- GPU�� ----------------\n");
		for(int i=0;i<Num;i++){
			printf("%d\t",arr[i]);
		}

		for(int i=0;i<Num-1;i++)
		for(int j=0;j<Num-1;j++){
			if(arr1[i]<arr1[j]){
				int t= arr1[i];
				arr1[i] = arr1[j];
				arr1[j] = t;
			}
		}

	 QueryPerformanceCounter(&nEndTime);

     utime=(double)(nEndTime.QuadPart-nBeginTime.QuadPart)/(double)nFreq.QuadPart;



		end1 =clock();

		printf("\n---------------- CPU�� ----------------\n");
		for(int i=0;i<Num-1;i++)
			printf("%d\t",arr1[i]);
		

		usetime = (double) (end1 - start1)*1000.0/CLK_TCK;


		printf("\n-------------- cpu: %f ms ----------------\n",utime);

		printf("\n-------------- gpu: %f ms-----------------\n\n",elapsedTime);


		hipEventDestroy(start);
		hipEventDestroy(stop);


		hipFree(ptr);
		return 0;
}